#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cu_mean_filter.h"
#include <stdio.h>
#include <stdint.h>
#define BLOCK_SIZE 16

__global__ void mean_filter_kernel(uint8_t *input, uint8_t *output,
                                   uint32_t width, uint8_t height,
                                   uint32_t window)
{
    uint32_t window_radis = window >> 1;
    uint32_t block_offset = BLOCK_SIZE - 2 * window_radis;
    __shared__ uint8_t shared_buf[BLOCK_SIZE][BLOCK_SIZE];
    uint32_t bx = blockIdx.x;
    uint32_t by = blockIdx.y;
    uint32_t tx = threadIdx.x;
    uint32_t ty = threadIdx.y;
    uint32_t input_idx = bx * block_offset + tx + (by * block_offset + ty) * width;
    shared_buf[ty][tx] = input[input_idx];
    // Synchronize to make sure the matrices are loaded
    __syncthreads();
    
    if (tx < window_radis ||
        tx >= (BLOCK_SIZE - window_radis) ||
        ty < window_radis ||
        ty >= (BLOCK_SIZE - window_radis))
    {
        return;
    }

    uint32_t i, j, sum = 0;
    uint8_t *ptr_filter_adder = &shared_buf[ty - window_radis][tx - window_radis];
    for (i = 0; i < window; i++)
    {
        for (j = 0; j < window; j++)
        {
            uint8_t adder = ptr_filter_adder[i * BLOCK_SIZE + j];
            sum += *ptr_filter_adder;
        }
    }
    output[input_idx] = sum / (window * window);
}

int32_t cuda_mean_filter(uint8_t *input, uint8_t *output,
                         uint32_t width, uint32_t height,
                         uint32_t window)
{
    uint8_t *cu_input_buf = NULL;
    uint8_t *cu_output_buf = NULL;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return -1;
    }

    cudaStatus = hipMalloc((void**)&cu_input_buf, width * height * sizeof(uint8_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return -1;
    }

    cudaStatus = hipMalloc((void**)&cu_output_buf, width * height * sizeof(uint8_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return -1;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(cu_input_buf, input, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return -1;
    }

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    uint32_t window_radis = window >> 1;
    uint32_t block_dim_x = (width + (BLOCK_SIZE - 2 * window_radis)) / (BLOCK_SIZE - 2 * window_radis);
    uint32_t block_dim_y = (height + (BLOCK_SIZE - 2 * window_radis)) / (BLOCK_SIZE - 2 * window_radis);
    dim3 grid(block_dim_x, block_dim_y);

    mean_filter_kernel<<<grid, threads>>>(cu_input_buf, cu_output_buf, width, height, window);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return -1;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, cu_output_buf, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return -1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return -1;
    }

    return 0;
}

#if 0
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
#endif